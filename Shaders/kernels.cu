#include "hip/hip_runtime.h"
#pragma once
#include "common.cuh"
#include "tonemapping.cuh"
//���ڶ�֡�ϳɵĺ˺���

extern "C" __global__ void AccumulateFrame(uint PixelCount, uint64 FrameCounter, uchar4 * OutputSRGBBuffer, float3 * IndirectOutputBuffer, float3 * AccumulateBuffer) {
	uint Idx=threadIdx.x + blockDim.x * blockIdx.x;
	if (Idx >= PixelCount) { return; }
	//֡��������ʾǰ������Ⱦ��N֡
	float3 AccumulatedColor = FrameCounter * fmaxf(make_float3(0), AccumulateBuffer[Idx]) + fmaxf(make_float3(0), IndirectOutputBuffer[Idx]);
	// float brightness = length(AccumulatedColor);
	// if (brightness < 0.0f) {
	// 	AccumulatedColor = make_float3(0, 0, 1);
	// }
	// else if (brightness > 1e6f) {
	// 	AccumulatedColor = make_float3(1, 0, 0);
	// }
	AccumulateBuffer[Idx] = AccumulatedColor / (FrameCounter + 1);
	OutputSRGBBuffer[Idx] = make_color(ACESFilm(AccumulatedColor / (FrameCounter + 1)));
}